
#include <hip/hip_runtime.h>
__global__ void matmul(int n, const float *A, const float *B, float *C){

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int bx = blockIdx.x;
  int by = blockIdx.y;

  int row = by*blockDim.y + ty;
  int col = bx*blockDim.x + tx;

  if(row < n && col < n){
    float val = 0.0;
    for(int i=0; i<n; ++i){
      val += A[row*n + i]*B[n*i + col];
    }
    C[row*n + col] = val;
  }
}

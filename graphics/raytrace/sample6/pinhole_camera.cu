#include "hip/hip_runtime.h"
#include <optix_world.h>
#include "helpers.h"

#include "RayTraceConfigInc.h"
#if RAYTRACE_CURAND
#include <hiprand/hiprand_kernel.h>
#endif


using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float  importance;
  int    depth;
};

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<uchar4, 2>              output_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );

rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;


#if RAYTRACE_CURAND
rtBuffer<hiprandState, 1> rng_states ;
#endif


RT_PROGRAM void pinhole_camera()
{
#if RAYTRACE_TIMEVIEW
  clock_t t0 = clock(); 
#endif
  // pixel coordinates into  [ -1 : 1, -1 : 1 ]
  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);
  
#if RAYTRACE_CURAND
  unsigned long long id = launch_index.x + launch_dim.x * launch_index.y ; 
  hiprandState rng = rng_states[id];
#endif

  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);


#if RAYTRACE_CURAND
  float u = hiprand_uniform(&rng); 
  prd.result.x = u ; 
  rng_states[id] = rng ; 
#endif


#if RAYTRACE_TIMEVIEW
  clock_t t1 = clock(); 
 
  float expected_fps   = 1.0f;
  float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
  output_buffer[launch_index] = make_color( make_float3(  pixel_time ) ); 
#else
  output_buffer[launch_index] = make_color( prd.result );
  //output_buffer[launch_index] = make_color(make_float3(0.5f));   // plain grey screen, not silhouette : all pixels go this way 
#endif
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_color( bad_color );
}






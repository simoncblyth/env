#include "hip/hip_runtime.h"
//usr/local/env/cuda/optix/OppositeRenderer/OppositeRenderer/RenderEngine/geometry_instance/TriangleMesh.cu
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.

*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtBuffer<float3> vertexBuffer;     
rtBuffer<float3> normalBuffer;
rtBuffer<float3> tangentBuffer;
rtBuffer<float3> bitangentBuffer;
rtBuffer<float2> texCoordBuffer;
rtBuffer<int3> indexBuffer; 
rtDeclareVariable(unsigned int, hasTangentsAndBitangents, ,);

rtDeclareVariable(float2, textureCoordinate, attribute textureCoordinate, ); 
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, ); 
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 
rtDeclareVariable(float3, tangent, attribute tangent, ); 
rtDeclareVariable(float3, bitangent, attribute bitangent, ); 

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

RT_PROGRAM void mesh_intersect(int primIdx)
{
    int3 index = indexBuffer[primIdx];

    float3 p0 = vertexBuffer[index.x];
    float3 p1 = vertexBuffer[index.y];
    float3 p2 = vertexBuffer[index.z];

    // Intersect ray with triangle
    float3 n;
    float  t, beta, gamma;
    if(intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma))
    {
        if(rtPotentialIntersection( t ))
        {
            if ( normalBuffer.size() == 0 )
            {
                shadingNormal = normalize( n );
            }
            else
            {
                float3 n0 = normalBuffer[index.x];
                float3 n1 = normalBuffer[index.y];
                float3 n2 = normalBuffer[index.z];
                shadingNormal = normalize( n1*beta + n2*gamma + n0*(1.0f-beta-gamma) );

                if(hasTangentsAndBitangents)
                {
                    float3 t0 = tangentBuffer[index.x];
                    float3 t1 = tangentBuffer[index.y];
                    float3 t2 = tangentBuffer[index.z];
                    tangent = normalize( t1*beta + t2*gamma + t0*(1.0f-beta-gamma) );

                    float3 b0 = bitangentBuffer[index.x];
                    float3 b1 = bitangentBuffer[index.y];
                    float3 b2 = bitangentBuffer[index.z];
                    bitangent = normalize( b1*beta + b2*gamma + b0*(1.0f-beta-gamma) );
                }
            }

            geometricNormal = normalize(n);

            // Texture UV coordinates

            if (texCoordBuffer.size() == 0)
            {
                textureCoordinate = make_float2( 0.0f );
            }
            else
            {
                float2 t0 = texCoordBuffer[index.x];
                float2 t1 = texCoordBuffer[index.y];
                float2 t2 = texCoordBuffer[index.z];
                textureCoordinate = t1*beta + t2*gamma + t0*(1.0f-beta-gamma);
            }

            rtReportIntersection(0);
        }
    }
}

RT_PROGRAM void mesh_bounds (int primIdx, float result[6])
{  
    const int3 index = indexBuffer[primIdx];

    const float3 v0   = vertexBuffer[ index.x ];
    const float3 v1   = vertexBuffer[ index.y ];
    const float3 v2   = vertexBuffer[ index.z ];
    const float  area = length(cross(v1-v0, v2-v0));

    optix::Aabb* aabb = (optix::Aabb*)result;

    if(area > 0.0f && !isinf(area))
    {
        aabb->m_min = fminf( fminf( v0, v1), v2 );
        aabb->m_max = fmaxf( fmaxf( v0, v1), v2 );
    }
    else 
    {
        aabb->invalidate();
    }
}

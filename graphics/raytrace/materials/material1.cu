#include "hip/hip_runtime.h"
#include "materials.h"

// shadingNormal is set by the closest hit intersection program 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(unsigned int, node_index, attribute node_index, );

rtDeclareVariable(float3, contrast_color, , );


rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_touch, prd_touch, rtPayload, );

//
// 
RT_PROGRAM void closest_hit_radiance()
{
  //prd_radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal))*0.5f + 0.5f;
  //prd_radiance.result = make_float3(0.f);
  prd_radiance.result = contrast_color ; 
}


RT_PROGRAM void closest_hit_touch()
{
  prd_touch.result = contrast_color ; 
  prd_touch.node = node_index ; 
}



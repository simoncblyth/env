
// nvcc -arch=sm_30 hello.cu -run ; rm a.out
// https://stackoverflow.com/questions/8003166/usage-of-printf-in-cuda-4-0-compilation-error


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

__global__ void helloCUDA(float f)
{
  printf("Hello thread %d, float(f)=%f  FLT_EPSILON:%g  sqrt(FLT_EPSILON) %g \n", threadIdx.x, f, FLT_EPSILON, sqrt(FLT_EPSILON));
}

__global__ void helloCUDA(double f)
{
  printf("Hello thread %d, double(e)=%e (g)%g (E)%E (G)%G DBL_EPSILON %g  \n", threadIdx.x, f, f, f, f, DBL_EPSILON );
}



int main()
{
  helloCUDA<<<1, 5>>>(1.2345f);

  helloCUDA<<<1, 5>>>(1.2345);

  


  hipDeviceReset();
  return 0;
}

// nvcc -o /tmp/minimal minimal.cu

// note no cuda includes, somehow nvcc automates that ?


#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10

__global__ void add(int *a, int *b) 
{
    int i = blockIdx.x;
    if (i<N) b[i] = 2*a[i];
}


int main(int /*argc*/, char** /*argv*/) 
{
    int ha[N], hb[N];
    int *da, *db;

    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    for (int i = 0; i<N; ++i) ha[i] = i;

    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<N, 1>>>(da, db);

    hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i<N; ++i) printf("%d\n", hb[i]);

    hipFree(da);
    hipFree(db);
    return 0;
}

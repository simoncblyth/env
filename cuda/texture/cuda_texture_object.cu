
#include <hip/hip_runtime.h>
/*
    http://devblogs.nvidia.com/parallelforall/cuda-pro-tip-kepler-texture-objects-improve-performance-and-flexibility/

    Kepler GPUs and CUDA 5.0 introduce a new feature called texture objects
    (sometimes called bindless textures, since they don’t require manual
    binding/unbinding) that greatly improves the usability and programmability of
    textures. Texture objects use the new cudaTextureObject_t class API, whereby
    textures become first-class C++ objects and can be passed as arguments just as
    if they were pointers.  There is no need to know at compile time which textures
    will be used at run time, which enables much more dynamic execution and
    flexible programming, as shown in the following code.

    Need to compile with at least compute capability 3.0 ie with:   -arch=sm_30 

*/

#define N 1024

// texture object is a kernel argument
__global__ void kernel(hipTextureObject_t tex) {
  int i = blockIdx.x *blockDim.x + threadIdx.x;
  float x = tex1Dfetch<float>(tex, i);
  // do some work using x ...
}

void call_kernel(hipTextureObject_t tex) {
  dim3 block(128,1,1);
  dim3 grid(N/block.x,1,1);
  kernel <<<grid, block>>>(tex);
}

int main() {
  // declare and allocate memory
  float *buffer;
  hipMalloc(&buffer, N*sizeof(float));

  // create texture object
  hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeLinear;
  resDesc.res.linear.devPtr = buffer;
  resDesc.res.linear.desc.f = hipChannelFormatKindFloat;
  resDesc.res.linear.desc.x = 32; // bits per channel
  resDesc.res.linear.sizeInBytes = N*sizeof(float);

  hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.readMode = hipReadModeElementType;

  // create texture object: we only have to do this once!
  hipTextureObject_t tex=0;
  hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

  call_kernel(tex); // pass texture as argument

  // destroy texture object
  hipDestroyTextureObject(tex);

  hipFree(buffer);
}
